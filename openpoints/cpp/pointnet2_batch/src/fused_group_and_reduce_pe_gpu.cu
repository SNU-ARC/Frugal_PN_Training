#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"
#include "fused_group_and_reduce_pe_gpu.h"


__global__ void fused_group_and_reduce_pe_grad_kernel_fast(int b, int c, int n, int npoints,
    const float *__restrict__ grad_out, const int *__restrict__ source_idx, const int *__restrict__ source_idx_pe, float *__restrict__ grad_points, float *__restrict__ grad_pe) {
    // grad_out: (B, C, npoints)
    // source_idx: (B, C, npoints)
    // source_idx_pe: (B, C, npoints)
    // output:
    //      grad_points: (B, C, N)
    //      grad_pe: (B, C, npoints, nsample)
    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b || c_idx >= c || pt_idx >= npoints) return;

    int output_idx = bs_idx * c * npoints + c_idx * npoints + pt_idx;
    grad_out += output_idx;
    source_idx += output_idx;
    source_idx_pe += output_idx;
 
    float grad = grad_out[0];
    atomicAdd(grad_points + source_idx[0] , grad);  // grad_points update
    grad_pe[source_idx_pe[0]] = grad;
}

void fused_group_and_reduce_pe_grad_kernel_launcher_fast(int b, int c, int n, int npoints,
    const float *grad_out, const int *source_idx, const int *source_idx_pe, float *grad_points, float *grad_pe) {
    // grad_out: (B, C, npoints)
    // source_idx: (B, C, npoints)
    // source_idx_pe: (B, C, npoints)
    // output:
    //      grad_points: (B, C, N)
    //      grad_pe: (B, C, npoints, nsample)
    hipError_t err;
    dim3 blocks(DIVUP(npoints, THREADS_PER_BLOCK), c, b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    fused_group_and_reduce_pe_grad_kernel_fast<<<blocks, threads, 0>>>(b, c, n, npoints, grad_out, source_idx, source_idx_pe, grad_points, grad_pe);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


__global__ void fused_group_and_reduce_pe_kernel_fast(int b, int c, int n, int npoints, int nsample, 
    const float *__restrict__ points, const int *__restrict__ idx, float *__restrict__ pe, float *__restrict__ out, int *__restrict__ source_idx, int *__restrict__ source_idx_pe) {
    // points: (B, C, N)
    // pe: (B, C, npoints, nsample)
    // idx: (B, npoints, nsample)
    // output:
    //      out: (B, C, npoints)
    //      source_idx: (B, C, npoints)
    //      source_idx_pe: (B, C, npoints)
    __shared__ float sdata[THREADS_PER_BLOCK/2];
    __shared__ int sidx[THREADS_PER_BLOCK/2];
    __shared__ int sidxpe[THREADS_PER_BLOCK/2];

    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int pt_idx = index / (nsample/2);
    if (bs_idx >= b || c_idx >= c || pt_idx >= npoints) return;

    int sample_idx = index % (nsample/2);

    // Gather input data to the shared mem.
    int smem_out_idx = threadIdx.x;
    const int *idx1 = idx + bs_idx * npoints * nsample + pt_idx * nsample + sample_idx;
    const int *idx2 = idx + bs_idx * npoints * nsample + pt_idx * nsample + sample_idx + nsample/2;
    int pe_idx1 = bs_idx * c * npoints * nsample + c_idx * npoints * nsample + pt_idx * nsample + sample_idx;
    int pe_idx2 = bs_idx * c * npoints * nsample + c_idx * npoints * nsample + pt_idx * nsample + sample_idx + nsample/2;
    int smem_in_idx1 = bs_idx * c * n + c_idx * n + idx1[0];
    int smem_in_idx2 = bs_idx * c * n + c_idx * n + idx2[0];

    float val1 = points[smem_in_idx1] + pe[pe_idx1];
    float val2 = points[smem_in_idx2] + pe[pe_idx2];
    float result = fmaxf(val1, val2);
    sdata[smem_out_idx] = result;
    sidx[smem_out_idx] = (result == val1) ? (smem_in_idx1) : (smem_in_idx2);
    sidxpe[smem_out_idx] = (result == val1) ? (pe_idx1) : (pe_idx2);
    __syncthreads();

    // Perform reduction.
    for (int s = nsample/4; s > 32; s>>=1) {
        if (sample_idx < s) {
            float val1 = sdata[smem_out_idx];
            float val2 = sdata[smem_out_idx + s];
            float result = fmaxf(val1, val2);
            sdata[smem_out_idx] = result;
            sidx[smem_out_idx] = (result == val1) ? (sidx[smem_out_idx]) : (sidx[smem_out_idx + s]);
            sidxpe[smem_out_idx] = (result == val1) ? (sidxpe[smem_out_idx]) : (sidxpe[smem_out_idx + s]);
        }
        __syncthreads();
    }
    for (int s = MIN(nsample/4, 32); s > 0; s>>=1) {
        if (sample_idx < s) {
            float val1 = sdata[smem_out_idx];
            float val2 = sdata[smem_out_idx + s];
            float result = fmaxf(val1, val2);
            sdata[smem_out_idx] = result;
            sidx[smem_out_idx] = (result == val1) ? (sidx[smem_out_idx]) : (sidx[smem_out_idx + s]);
            sidxpe[smem_out_idx] = (result == val1) ? (sidxpe[smem_out_idx]) : (sidxpe[smem_out_idx + s]);
        }
    }

    int in_idx = smem_out_idx;
    int out_idx = bs_idx * c * npoints + c_idx * npoints + pt_idx;

    if (sample_idx == 0) {
        out[out_idx] = sdata[in_idx];
        source_idx[out_idx] = sidx[in_idx];
        source_idx_pe[out_idx] = sidxpe[in_idx];
    }
}


void fused_group_and_reduce_pe_kernel_launcher_fast(int b, int c, int n, int npoints, int nsample, 
    const float *points, const int *idx, float *pe, float *out, int *source_idx, int *source_idx_pe) {
    // points: (B, C, N)
    // pe: (B, C, npoints, nsample)
    // idx: (B, npoints, nsample)
    // output:
    //      out: (B, C, npoints)
    //      source_idx: (B, C, npoints)
    //      source_idx_pe: (B, C, npoints)
    hipError_t err;
    int blocksize = MAX(nsample, THREADS_PER_BLOCK);
    dim3 blocks(DIVUP(npoints * nsample, blocksize), c, b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(blocksize/2);

    fused_group_and_reduce_pe_kernel_fast<<<blocks, threads, 0>>>(b, c, n, npoints, nsample, points, idx, pe, out, source_idx, source_idx_pe);

    // hipDeviceSynchronize();  // for using printf in kernel function
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


